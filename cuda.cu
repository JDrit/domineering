#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>

#define THREADS_PER_BLOCK 1024

#define CUDA_ERROR_CHECK
#define CUDA_SAFE_CALL( err ) __CUDA_SAFE_CALL( err, __FILE__, __LINE__ )
#define CUDA_CHECK_ERROR()    __CUDA_CHECK_ERROR( __FILE__, __LINE__ )
#define GET_INDEX(y_max, x, y) (y_max * x + y)
#define LOG_PRINTF(...) do { \
    time_t ltime = time(NULL); \
    printf("%.24s: ",asctime( localtime(&ltime))); \
    printf(__VA_ARGS__); \
} while (0) 
#define LOG_FPRINTF(f, ...) do { \
    time_t ltime = time(NULL); \
    fprintf(f, "%.24s: ",asctime( localtime(&ltime))); \
    fprintf(f, __VA_ARGS__); \
} while (0) 

using namespace std;

typedef struct {
    uint64_t bitboards[2];
} Board;

void outOfMemHandler() {
    LOG_FPRINTF(stderr, "Unable to satisfy request for memory\n");
    std::abort();
}

inline void __CUDA_SAFE_CALL( hipError_t err, const char *file, const int line ) {
    #ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err ) {
        LOG_FPRINTF(stderr, "CUDA_SAFE_CALL() failed at %s:%i : %s\n", file, 
                line, hipGetErrorString( err ) );
        exit( -1 );
    }
    #endif
}

inline void __CUDA_CHECK_ERROR( const char *file, const int line ) {
    #ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err ) {
        LOG_FPRINTF(stderr, "CUDA_CHECK_ERROR() failed at %s:%i : %s\n", file, 
                line, hipGetErrorString(err));
        exit( -1 );
    }
    
    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err ) {
        LOG_FPRINTF( stderr, "CUDA_CHECK_ERROR() with sync failed at %s:%i : %s\n", 
                file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
    #endif
}

inline bool boards_equal(Board *b1, Board *b2) {
    return b1->bitboards[0] == b2->bitboards[0] && b1->bitboards[1] == b2->bitboards[1];
}

int compare_boards(const void *v1, const void *v2) {
    Board *b1 = (Board*) v1;
    Board *b2 = (Board*) v2;
    return b1->bitboards[0] < b2->bitboards[0];
}

__host__ __device__ inline bool get_location(Board *board, int y_max, int x, int y) {
    //TODO this will probs break for bigger boards
    int index = GET_INDEX(y_max, x, y);
    int boardIndex;
    double offset;
    
    if (index < 63) { // board 1
        boardIndex = 0;
        offset = pow(2.0, index + 1);
    } else { // board 2
        boardIndex = 1;
        offset = pow(2.0, index - 63);
    } 
    return (board->bitboards[boardIndex] & (uint64_t) offset) != 0;
}

__host__ __device__ inline void set_location(Board *board, int y_max, int x, int y) {
    //TODO this will probs break for bigger boards
    int index = GET_INDEX(y_max, x, y);
    int boardIndex;
    uint64_t offset;

    if (index < 63) { // board 1
        boardIndex = 0;
        offset = pow(2.0, index + 1);
    } else { // board 2
        boardIndex = 1;
        offset = pow(2.0, index - 63);
    } 
    board->bitboards[boardIndex] = board->bitboards[boardIndex] | offset;
}


__host__ __device__ inline bool is_valid(Board *board) {
    return board->bitboards[0] & 1 != 0;
}

__host__ __device__ inline void set_valid(Board *board, bool valid) {
    if (valid) {
        board->bitboards[0] = board->bitboards[0] | 1;    
    } else {
        board->bitboards[0] = board->bitboards[0] & 0;
    }
}

__host__ __device__ void print_board(Board *board, int x_max, int y_max) {
    printf("size: (%d, %d)\n", x_max, y_max);
    printf("   ");
    for (int y = 0 ; y < y_max ; y++) {
        printf("%d  ", y);
    }
    printf("\n");
    for (int x = 0 ; x < x_max ; x++) {
        printf("%d ", x);
        for (int y = 0 ; y < y_max ; y++) {
            if (get_location(board, y_max, x, y) == true) {
                printf(" X ");
            } else {
                printf(" . ");
            }
        }
        printf("\n");
    }
}

__device__ inline void copy_left(Board *board, int x_max, int y_max) {
    int middle = y_max / 2;
    int leftCount = 0;
    int rightCount = 0;
    if (y_max % 2 == 0) {
        for (int x = 0 ; x < x_max ; x++) {
            for (int y = 0 ; y < middle ; y++) {
                if (get_location(board, y_max, x, y) == true) {
                    leftCount++;
                }
            }

            for (int y = middle ; y < y_max ; y++) {
                if (get_location(board, y_max, x, y) == true) {
                    rightCount++;
                }
            }
        }
    } else {
        for (int x = 0 ; x < x_max ; x++) {
            for (int y = 0 ; y < middle ; y++) {
                if (get_location(board, y_max, x, y) == true) {
                    leftCount++;
                }
            }

            for (int y = middle + 1 ; y < y_max ; y++) {
                if (get_location(board, y_max, x, y) == true) {
                    rightCount++;
                }
            }
        }
    }
    if (leftCount < rightCount) {
        Board *tmpBoard = new Board; 
        memcpy(tmpBoard, board, sizeof(Board));
        memset(board, 0, sizeof(Board));
        set_valid(board, true);
        for (int x = 0 ; x < x_max ; x++) {
            for (int y = 0 ; y < y_max ; y++) {
                if (get_location(tmpBoard, y_max, x, y) == true) {
                    set_location(board, y_max, x, y_max - y - 1);
                } 
            }
        }
        delete tmpBoard;
    }
}

// blockIdx.x  = block index within the grid
// blockDim.x  = dimension of the block
// threadIdx.x = thread index within the block

__global__ void next_boards(Board *input, Board *output, int branching, 
        int x_max, int y_max, bool vertical) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    Board board = input[index];

    int count = 0; 
    if (is_valid(&board)) { 
        if (vertical) {
            for (int x = 0 ; x < x_max - 1; x++) {
                for (int y = 0 ; y < y_max ; y++) {
                    if (!get_location(&board, y_max, x, y) && 
                            !get_location(&board, y_max, x + 1, y)) {
                        memcpy(&output[index * branching + count], &board, sizeof(Board));
                        set_location(&output[index * branching + count], y_max, x, y);
                        set_location(&output[index * branching + count], y_max, x + 1, y); 
                        count++;
                    }
                }
            }
        } else {
            for (int x = 0 ; x < x_max ; x++) {
                for (int y = 0 ; y < y_max - 1 ; y++) {
                    if (!get_location(&board, y_max, x, y) && 
                            !get_location(&board, y_max, x, y + 1)) {
                        memcpy(&output[index * branching + count], &board, sizeof(Board));
                        set_location(&output[index * branching + count], y_max, x, y);
                        set_location(&output[index * branching + count], y_max, x, y + 1);
                        count++;
                    }
                }
            }
        }
    }
    for (int i = 0 ; i < count ; i++) {
        Board *board = &output[index * branching + i];
        set_valid(board, true);
        copy_left(board, x_max, y_max);
    }
    for (int i = count ; i < branching ; i++) {
        set_valid(&output[index * branching + i], false);
    }

    //__syncthreads();
}

int best = 0;

void work_down(Board* input, int x_max, int y_max, int inCount, bool vertical, int depth) {
    if (depth > best) {
        best = depth;
    }
    LOG_PRINTF("\n");
    if (inCount == 0) {
        for (int i = 0 ; i < depth ; i++)
            LOG_PRINTF(" ");
        LOG_PRINTF("no more moves at depth: %d\n", depth);
        return;
    }
    
    LOG_PRINTF("starting for for depth: %d\n", depth);
    Board *dev_input;
    Board *dev_output;

    int inputSize = inCount * sizeof(Board);

    //TODO might be wrong branching count
    int branching = x_max * y_max;
    int outCount = inCount * branching;
    int outputSize = outCount * sizeof(Board);

    
    LOG_PRINTF("best            : %d\n", best);
    
    LOG_PRINTF("input count     : %d\n", inCount);
    
    LOG_PRINTF("branching count : %d\n", branching);
    
    LOG_PRINTF("output count    : %d\n", outCount);

    Board *output = new Board[outCount];
    
    CUDA_SAFE_CALL(hipMalloc((void **) &dev_input, inputSize));
    CUDA_SAFE_CALL(hipMalloc((void **) &dev_output, outputSize));
    CUDA_SAFE_CALL(hipMemcpy(dev_input, input, inputSize, hipMemcpyHostToDevice));
    
    int blocks = inCount / THREADS_PER_BLOCK;
    blocks = (blocks == 0) ? 1 : blocks;
    
    next_boards<<<blocks, THREADS_PER_BLOCK>>>(dev_input, dev_output, branching, 
            x_max, y_max, vertical);
    CUDA_CHECK_ERROR();
    
    CUDA_SAFE_CALL(hipMemcpy(output, dev_output, outputSize, hipMemcpyDeviceToHost));

    
    LOG_PRINTF("gpu done...\n");

    bool next = false;
    for (int i = 0 ; i < outCount ; i++ ) {
        Board board = (Board) output[i];
        if (is_valid(&board) == true) {
            next = true;
            break;
        }
    }
    
    CUDA_SAFE_CALL(hipFree(dev_input));
    CUDA_SAFE_CALL(hipFree(dev_output));
    
    if (next) {
        
        //TODO fix performance of this section
        
        int validCount = 0;
        Board *validOutput = new Board[outCount];
        for (int i = 0 ; i < outCount ; i++) {
            if (is_valid(&output[i]) == true) {
                memcpy(&validOutput[validCount++], &output[i], sizeof(Board));
            }
        }

        // sorts the new output so that duplicates can be removed
        qsort(validOutput, validCount, sizeof(Board), compare_boards);
        Board *noDuplicates = new Board[outCount];
        int dupCount = 1;

        Board last = validOutput[0];
        memcpy(&noDuplicates[0], &validOutput[0], sizeof(Board));
        
        for (int i = 1 ; i < validCount ; i++) {
            if (!boards_equal(&last, &validOutput[i])) {
                memcpy(&noDuplicates[dupCount++], &validOutput[i], sizeof(Board));
                last = validOutput[i];
            }
        }
        
        LOG_PRINTF("valid count     : %d\n", validCount);
        
        LOG_PRINTF("duplicate count : %d\n", dupCount);
        delete[] output;
        delete[] validOutput;

        int size = 3000000;
        if (dupCount > size) {
            LOG_PRINTF("\n");
            
            LOG_PRINTF("splitting...\n");

            for (int i = 0 ; i < dupCount ; i += size) {
                if (dupCount < i + size) {
                    work_down(noDuplicates + i, x_max, y_max, dupCount - i, !vertical, depth + 1);
                } else {
                    work_down(noDuplicates + i, x_max, y_max, size, !vertical, depth + 1);
                }
            }
        } else {
            work_down(noDuplicates, x_max, y_max, dupCount, !vertical, depth + 1);
        }
        delete[] noDuplicates;
    } else { 
        LOG_PRINTF("no more moves\n");
        delete[] output;
    }
}


// main routine that executes on the host
int main(void) {
    unsigned char x = 6;
    unsigned char y = 6;

    std::set_new_handler(outOfMemHandler);

    LOG_PRINTF("remember to kill the X session\n");
    LOG_PRINTF("Board size: %d\n", sizeof(Board));

    int inCount = 1;
    Board *inputBoards = new Board[inCount];
    memset(&inputBoards[0], 0, sizeof(Board));
    set_valid(&inputBoards[0], true);
    LOG_PRINTF("initial\n");
    print_board(&inputBoards[0], x, y);
    work_down(inputBoards, x, y, 1, true, 0);
    delete[] inputBoards; 
    return 0;
}
