#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_ERROR_CHECK
#define CUDA_SAFE_CALL( err ) __CUDA_SAFE_CALL( err, __FILE__, __LINE__ )
#define CUDA_CHECK_ERROR()    __CUDA_CHECK_ERROR( __FILE__, __LINE__ )
#define GET_INDEX(y_max, x, y) (y_max * x + y)

using namespace std;

typedef struct {
    uint64_t bitboards[2];
} Board;

void outOfMemHandler() {
    std::cerr << "Unable to satisfy request for memory\n";
    std::abort();
}

inline void __CUDA_SAFE_CALL( hipError_t err, const char *file, const int line ) {
    #ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err ) {
        fprintf( stderr, "CUDA_SAFE_CALL() failed at %s:%i : %s\n", file, 
                line, hipGetErrorString( err ) );
        exit( -1 );
    }
    #endif
}

inline void __CUDA_CHECK_ERROR( const char *file, const int line ) {
    #ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err ) {
        fprintf( stderr, "CUDA_CHECK_ERROR() failed at %s:%i : %s\n", file, 
                line, hipGetErrorString( err ) );
        exit( -1 );
    }
    
    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err ) {
        fprintf( stderr, "CUDA_CHECK_ERROR() with sync failed at %s:%i : %s\n", 
                file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
    #endif
}

inline bool boards_equal(Board *b1, Board *b2) {
    return b1->bitboards[0] == b2->bitboards[0] && b1->bitboards[1] == b2->bitboards[1];
}

int compare_boards(const void *v1, const void *v2) {
    Board *b1 = (Board*) v1;
    Board *b2 = (Board*) v2;
    return b1->bitboards[0] < b2->bitboards[0];
}

__host__ __device__ inline bool get_location(Board *board, int y_max, int x, int y) {
    //TODO this will probs break for bigger boards
    int index = GET_INDEX(y_max, x, y);
    int boardIndex;
    double offset;
    
    if (index < 63) { // board 1
        boardIndex = 0;
        offset = pow(2.0, index + 1);
    } else { // board 2
        boardIndex = 1;
        offset = pow(2.0, index - 63);
    } 
    return (board->bitboards[boardIndex] & (int) offset) != 0;
}

__device__ inline void set_location(Board *board, int y_max, int x, int y) {
    //TODO this will probs break for bigger boards
    int index = GET_INDEX(y_max, x, y);
    int boardIndex;
    int offset;

    if (index < 63) { // board 1
        boardIndex = 0;
        offset = pow(2.0, index + 1);
    } else { // board 2
        boardIndex = 1;
        offset = pow(2.0, index - 63);
    } 
    board->bitboards[boardIndex] = board->bitboards[boardIndex] | offset;
}


__host__ __device__ inline bool is_valid(Board *board) {
    return board->bitboards[0] & 1 != 0;
}

__host__ __device__ inline void set_valid(Board *board, bool valid) {
    if (valid) {
        board->bitboards[0] = board->bitboards[0] | 1;    
    } else {
        board->bitboards[0] = board->bitboards[0] & 0;
    }
}

__host__ __device__ void print_board(Board *board, int x_max, int y_max) {
    printf("size: (%d, %d)\n", x_max, y_max);
    printf("   ");
    for (int y = 0 ; y < y_max ; y++) {
        printf("%d  ", y);
    }
    printf("\n");
    for (int x = 0 ; x < x_max ; x++) {
        printf("%d ", x);
        for (int y = 0 ; y < y_max ; y++) {
            if (get_location(board, y_max, x, y) == true) {
                printf(" X ");
            } else {
                printf(" . ");
            }
        }
        printf("\n");
    }
}

// blockIdx.x  = block index within the grid
// blockDim.x  = dimension of the block
// threadIdx.x = thread index within the block

__global__ void next_boards(Board *input, Board *output, int branching, 
        int x_max, int y_max, bool vertical) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    Board board = input[index];
    int count = 0; 
    if (is_valid(&board)) { 
        if (vertical) {
            for (int x = 0 ; x < x_max - 1; x++) {
                for (int y = 0 ; y < y_max ; y++) {
                    if (!get_location(&board, y_max, x, y) && 
                            !get_location(&board, y_max, x + 1, y)) {
                        memcpy(&output[index * branching + count], &board, sizeof(Board));
                        set_location(&output[index * branching + count], y_max, x, y);
                        set_location(&output[index * branching + count], y_max, x + 1, y); 
                        count++;
                    }
                }
            }
        } else {
            for (int x = 0 ; x < x_max ; x++) {
                for (int y = 0 ; y < y_max - 1 ; y++) {
                    if (!get_location(&board, y_max, x, y) && 
                            !get_location(&board, y_max, x, y + 1)) {
                        memcpy(&output[index * branching + count], &board, sizeof(Board));
                        set_location(&output[index * branching + count], y_max, x, y);
                        set_location(&output[index * branching + count], y_max, x, y + 1);
                        count++;
                    }
                }
            }
        }
    }
    for (int i = 0 ; i < count ; i++) {
        set_valid(&output[index * branching + i], true);
    }
    for (int i = count ; i < branching ; i++) {
        set_valid(&output[index * branching + i], false);
    }
}

void work_down(Board* input, int x_max, int y_max, int inCount, bool vertical, int depth) {
    if (inCount == 0) {
        printf("\nno more moves at depth: %d\n", depth);
        return;
    }
    printf("\nstarting for for depth: %d\n", depth);
    Board *dev_input;
    Board *dev_output;

    int inputSize = inCount * sizeof(Board);

    //TODO might be wrong branching count
    int branching = x_max * y_max - 2 * depth;
    int outCount = inCount * branching;
    int outputSize = outCount * sizeof(Board);

    printf("input count     : %d\n", inCount);
    printf("branching count : %d\n", branching);
    printf("output count    : %d\n", outCount);

    Board *output = new Board[outCount];

    CUDA_SAFE_CALL(hipMalloc((void **) &dev_input, inputSize));
    CUDA_SAFE_CALL(hipMalloc((void **) &dev_output, outputSize));
    
    CUDA_SAFE_CALL(hipMemcpy(dev_input, input, inputSize, hipMemcpyHostToDevice));
    
    next_boards<<<inCount, 1>>>(dev_input, dev_output, branching, x_max, y_max, vertical);
    CUDA_CHECK_ERROR();
    
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    CUDA_SAFE_CALL(hipMemcpy(output, dev_output, outputSize, hipMemcpyDeviceToHost));

    bool next = false;
    for (int i = 0 ; i < outCount ; i++ ) {
        Board board = (Board) output[i];
        if (is_valid(&board) == true) {
            next = true;
            break;
        }
    }
    
    CUDA_SAFE_CALL(hipFree(dev_input));
    CUDA_SAFE_CALL(hipFree(dev_output));
    
    if (next) {
        int validCount = 0;
        Board *validOutput = new Board[outCount];
        for (int i = 0 ; i < outCount ; i++) {
            if (is_valid(&output[i]) == true) {
                memcpy(&validOutput[validCount++], &output[i], sizeof(Board));
            }
        }
        
        // sorts the new output so that duplicates can be removed
        qsort(validOutput, validCount, sizeof(Board), compare_boards);
        Board *noDuplicates = new Board[outCount];
        int dupCount = 1;

        Board last = validOutput[0];
        memcpy(&noDuplicates[0], &validOutput[0], sizeof(Board));
        
        for (int i = 1 ; i < validCount ; i++) {
            if (!boards_equal(&last, &validOutput[i])) {
                memcpy(&noDuplicates[dupCount++], &validOutput[i], sizeof(Board));
                last = validOutput[i];
            }
        }
        printf("valid count     : %d\n", validCount);
        printf("duplicate count : %d\n", dupCount);
        delete[] output;
        delete[] validOutput;
        work_down(noDuplicates, x_max, y_max, dupCount, !vertical, depth + 1);
        delete[] noDuplicates;
    } else {
        printf("no more moves\n");
        delete[] output;
    }
}

// main routine that executes on the host
int main(void) {
    unsigned char x = 30;
    unsigned char y = 2;

    std::set_new_handler(outOfMemHandler);

    printf("Board size: %d\n", sizeof(Board));

    int inCount = 1;
    Board *inputBoards = new Board[inCount];
    inputBoards[0].bitboards[0] = 0;
    inputBoards[0].bitboards[1] = 0;
    set_valid(&inputBoards[0], true);
    printf("initial\n");
    print_board(&inputBoards[0], x, y);
    work_down(inputBoards, x, y, 1, true, 0);
    delete[] inputBoards; 
    return 0;
}
