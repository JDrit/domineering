#include "hip/hip_runtime.h"
#include "domineering.h"

#define X_MAX 11
#define Y_MAX 4
#define MAX_SIZE 50000

#define NO_WINNER -1
#define NEXT_WIN 1
#define PREV_WIN 2

void outOfMemHandler() {
    LOG_FPRINTF(stderr, "Unable to satisfy request for memory\n");
    std::abort();
}

inline void __CUDA_SAFE_CALL( hipError_t err, const char *file, const int line ) {
    #ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err ) {
        LOG_FPRINTF(stderr, "CUDA_SAFE_CALL() failed at %s:%i : %s\n", file, 
                line, hipGetErrorString( err ) );
        exit( -1 );
    }
    #endif
}

inline void __CUDA_CHECK_ERROR( const char *file, const int line ) {
    #ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err ) {
        LOG_FPRINTF(stderr, "CUDA_CHECK_ERROR() failed at %s:%i : %s\n", file, 
                line, hipGetErrorString(err));
        exit( -1 );
    }
   #endif
}

__device__ double board_distance(Board *board) {
    double distance = 0;

    int middle_x = X_MAX / 2;
    int middle_y = Y_MAX / 2; 

    for (int x = 0 ; x < X_MAX ; x++) {
        for (int y = 0 ; y < Y_MAX ; y++) {
            if (get_location(board, Y_MAX, x, y) == true) {
                distance += sqrt(pow(1.0 * x - middle_x, 2.0) + pow(1.0 * y - middle_y, 2.0));
            }
        }
    }
    return distance;
}

// blockIdx.x  = block index within the grid
// blockDim.x  = dimension of the block
// threadIdx.x = thread index within the block

__global__ void next_boards(Board *input, Board *output, int branching, bool vertical, 
        int max_index) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int count = 0; 

    // makes sure that the threads only read the given input. This can happen
    // when the amount of threads per block do not line up with the total 
    // input count.
    if (index < max_index) {
        Board board = input[index];
        // only process valid board configurations
        if (is_valid(&board)) { 
            if (vertical) {
                for (int x = 0 ; x < X_MAX - 1; x++) {
                    for (int y = 0 ; y < Y_MAX ; y++) {
                        if (!get_location(&board, Y_MAX, x, y) && 
                                !get_location(&board, Y_MAX, x + 1, y)) {
                            memcpy(&output[index * branching + count], &board, sizeof(Board));
                            set_location(&output[index * branching + count], Y_MAX, x, y);
                            set_location(&output[index * branching + count], Y_MAX, x + 1, y); 
                            set_valid(&output[index * branching + count], true);
                            output[index * branching + count].parent = index;
                            count++;
                        }
                    }
                }
            } else {
                for (int x = 0 ; x < X_MAX ; x++) {
                    for (int y = 0 ; y < Y_MAX - 1 ; y++) {
                        if (!get_location(&board, Y_MAX, x, y) && 
                                !get_location(&board, Y_MAX, x, y + 1)) {
                            memcpy(&output[index * branching + count], &board, sizeof(Board));
                            set_location(&output[index * branching + count], Y_MAX, x, y);
                            set_location(&output[index * branching + count], Y_MAX, x, y + 1);
                            set_valid(&output[index * branching + count], true);
                            output[index * branching + count].parent = index;
                            count++;
                        }
                    }
                }
            }
        }
        for (int i = count ; i < branching ; i++) {
            Board *board = &output[index * branching + i];
            board->bitboards[0] = 0;
            board->bitboards[1] = 0;
        }
    }
}

__device__ bool vertical_equal(Board *b1, Board *b2) {
    Board tmp;
    memset(&tmp, 0, sizeof(Board));
    set_valid(&tmp, true);

    for (int x = 0 ; x < X_MAX ; x++) {
        for (int y = 0 ; y < Y_MAX ; y++) {
            if (get_location(b1, Y_MAX, x, y) == true) {
                set_location(&tmp, Y_MAX, x, Y_MAX - y - 1);
            }
        }
    }
    return tmp.bitboards[0] == b2->bitboards[0] && tmp.bitboards[1] == b2->bitboards[1];
}

__device__ bool horizontal_equal(Board *b1, Board *b2) {
    Board tmp;
    memset(&tmp, 0, sizeof(Board));
    set_valid(&tmp, true);

    for (int x = 0 ; x < X_MAX ; x++) {
        for (int y = 0 ; y < Y_MAX ; y++) {
            if (get_location(b1, Y_MAX, x, y) == true) {
                set_location(&tmp, Y_MAX, X_MAX  - x - 1, y);
            }
        }
    }
    return tmp.bitboards[0] == b2->bitboards[0] && tmp.bitboards[1] == b2->bitboards[1];
}


__device__ bool rotate_equal(Board *b1, Board *b2) {
    Board tmp;
    memset(&tmp, 0, sizeof(Board));
    set_valid(&tmp, true);

    for (int x = 0; x < X_MAX; x++) {
        for (int y = 0; y < Y_MAX; y++) {
            if (get_location(b1, Y_MAX, Y_MAX - y - 1, x) == true) {
                set_location(&tmp, Y_MAX, x, y);
            } 
        }
    }
    bool result = tmp.bitboards[0] == b2->bitboards[0] && tmp.bitboards[1] == b2->bitboards[1];
    if (result == true)
        return true;

    memset(&tmp, 0, sizeof(Board));
    set_valid(&tmp, true);

    for (int x = 0 ; x < X_MAX ; x++) {
        for (int y = 0 ; y < Y_MAX ; y++) {
            if (get_location(b1, Y_MAX, x, y) == true) {
                set_location(&tmp, Y_MAX, X_MAX - x - 1, Y_MAX - y - 1);
            }
        }
    }
    result = tmp.bitboards[0] == b2->bitboards[0] && tmp.bitboards[1] == b2->bitboards[1];
    if (result == true)
        return true;

    memset(&tmp, 0, sizeof(Board));
    set_valid(&tmp, true);

    for (int x = 0 ; x < X_MAX ; x++) {
        for (int y= 0 ; y < Y_MAX ; y++) {
            if (get_location(b1, Y_MAX, y, X_MAX - x -1) == true) {
                set_location(&tmp, Y_MAX, x, y);
            }
        }
    }

    return (tmp.bitboards[0] == b2->bitboards[0] && 
            tmp.bitboards[1] == b2->bitboards[1]) ||
        vertical_equal(&tmp, b2) || horizontal_equal(&tmp, b2);
}

__device__ bool operator ==(const Board& b1, const Board& b2) {
    Board board1 = b1;
    Board board2 = b2;
    return (b1.bitboards[0] == b2.bitboards[0] && b2.bitboards[1] == b2.bitboards[1]) ||
        rotate_equal(&board1, &board2) || vertical_equal(&board1, &board2) ||
        horizontal_equal(&board1, &board2);
}

int best = 0;

char* work_down(Board* input, int inCount, bool vertical, int depth) {
    best = max(best, depth);
    if (inCount == 0) {
        LOG_PRINTF("no more moves at depth: %d\n", depth);
        return NULL;
    }

    int branching = X_MAX * Y_MAX;
    int outCount = inCount * branching;
    
    Board *dev_boards;
    Board *dev_input;
    
    size_t totalSize = inCount * sizeof(Board) + outCount * sizeof(Board);

    printf("\n");
    LOG_PRINTF("depth           : %d\n", depth);
    LOG_PRINTF("max             : %d\n", best);
    LOG_PRINTF("input count     : %d\n", inCount);
    LOG_PRINTF("branching count : %d\n", branching);
    LOG_PRINTF("output count    : %d\n", outCount);
    LOG_PRINTF("mallocing size  : %zu\n", totalSize);
    
    CUDA_SAFE_CALL(hipMalloc((void**) &dev_input, inCount * sizeof(Board)));
    CUDA_SAFE_CALL(hipMemcpy(dev_input, input, inCount * sizeof(Board),
                hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMalloc((void **) &dev_boards, outCount * sizeof(Board)));

    int blocks = (int) ceil((inCount * 1.0) / THREADS_PER_BLOCK);
    next_boards<<<blocks, THREADS_PER_BLOCK>>>(dev_input, dev_boards, 
            branching, vertical, inCount);
    CUDA_CHECK_ERROR();
    CUDA_SAFE_CALL(hipFree(dev_input));
    
    size_t N = outCount;
    thrust::device_ptr<Board> d_board_ptr = thrust::device_pointer_cast(dev_boards);
    thrust::device_vector<Board> d_board_vec(d_board_ptr, d_board_ptr + N);

    // removes all invalid boards from the vector
    d_board_vec.erase(thrust::remove_if(d_board_vec.begin(), d_board_vec.end(),
                is_valid_struct()), d_board_vec.end());

    size_t size = d_board_vec.size();
    LOG_PRINTF("output size     : %d\n", size);
    Board *host_output = new Board[size];
    Board* dv_ptr = thrust::raw_pointer_cast(d_board_vec.data());
    CUDA_SAFE_CALL(hipMemcpy(host_output, dv_ptr, size * sizeof(Board), hipMemcpyDeviceToHost));

    d_board_vec.clear();
    d_board_vec.shrink_to_fit();
    CUDA_SAFE_CALL(hipFree(dev_boards));

    if (size == 0) {
        char *winners = new char[inCount];
        for (int i = 0 ; i < inCount ; i++) {
            winners[i] = 'P';
        }
        delete[] host_output;
        return winners;
    } else {
        char *nextWinners = work_down(host_output, size, !vertical, depth + 1);
        char *winners = new char[inCount];
        int offset = 0;
        for (int i = 0 ; i < inCount ; i++) {
            char winner = 'P';
            while (offset < size && host_output[offset].parent == i) {
                if (nextWinners[offset] == 'P') {
                    winner = 'N';
                }
                offset++;
            }
            winners[i] = winner;
        }
        delete[] nextWinners;
        delete[] host_output;
        return winners;
    }
    

    

}


// main routine that executes on the host
int main(void) {

    std::set_new_handler(outOfMemHandler);

    LOG_PRINTF("remember to kill the X session\n");
    LOG_PRINTF("Board size: %d\n", sizeof(Board));

    int inCount = 1;
    Board *inputBoards = new Board[inCount];
    memset(&inputBoards[0], 0, sizeof(Board));
    set_valid(&inputBoards[0], true);

    char *winner = work_down(inputBoards, 1, true, 0);
    printf("winner vertical first: %c\n", winner[0]);
    delete[] winner;
    delete[] inputBoards;
    return 0;
}
